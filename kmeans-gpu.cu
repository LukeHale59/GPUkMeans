#include "hip/hip_runtime.h"
// Implementation of the KMeans Algorithm
// reference: https://github.com/marcoscastro/kmeans

#include <iostream>
#include <vector>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <chrono>
#include <xmmintrin.h>
#include <limits>
#include "config_t.h"
#include <cassert>
#include <cstdlib>
#include <string>
#include <unistd.h>


using namespace std;

void parseargs(int argc, char** argv, config_t& cfg) {
    // parse the command-line options
    int opt;
    while ((opt = getopt(argc, argv, "k:s:t:p:h")) != -1) {
        switch (opt) {
          case 'k': cfg.clusters = atoi(optarg); break;
          case 's': cfg.seed = atoi(optarg); break;
		  case 't': cfg.threads = atoi(optarg); break;
		  case 'p': cfg.total_points = atoi(optarg); break;
		  case 'h': cfg.seed2 = atoi(optarg); break;
        }
    }
}

struct PointStruct {
    int id_cluster;
    double* values;

	PointStruct() : id_cluster(-1), values(nullptr) {}

    // Constructor to initialize values array with a specified size
    PointStruct(int size) : id_cluster(-1) {
        values = new double[size];
    }

    // Destructor to free memory allocated for values array
    ~PointStruct() {
        delete[] values;
    }
};

struct ClusterStruct {
    int numPoints;
    double* central_values;
    double* central_values_sums;

	ClusterStruct() : numPoints(0), central_values(nullptr), central_values_sums(nullptr) {}
    // Constructor to initialize central_values and central_values_sums arrays with a specified size
    ClusterStruct(int size) : numPoints(0){
        numPoints = 0;
        central_values = new double[size];
        central_values_sums = new double[size];
    }

    // Destructor to free memory allocated for central_values and central_values_sums arrays
    ~ClusterStruct() {
        delete[] central_values;
        delete[] central_values_sums;
    }

};

int main(int argc, char *argv[])
{
	// Parse command line arguments using getopt()
	config_t config;
    parseargs(argc, argv, config);
	//srand (time(NULL));
    srand (config.seed);

	int total_points, total_values, K = config.clusters, max_iterations, has_name;
	cin >> total_points >> total_values >> max_iterations >> has_name;

	if(config.total_points != -1){
		total_points = config.total_points;
	}
    
	PointStruct* points = (PointStruct*)malloc(total_points * sizeof(PointStruct));

	// Initialize each PointStruct object
	for(int i = 0; i < total_points; i++) {
	    // Initialize the PointStruct object with the size of values array
	    new (&points[i]) PointStruct(total_values);
	
	    // Fill in the values for each 'values' array
	    for(int j = 0; j < total_values; j++) {
	        double value;
	        cin >> value;
	        points[i].values[j] = value;
	    }
	}

	ClusterStruct* clusters = (ClusterStruct*)malloc(K * sizeof(ClusterStruct(total_values)));
	std::chrono::high_resolution_clock::time_point begin = chrono::high_resolution_clock::now();
    
	if(K > total_points)
		return -1;
	vector<int> prohibited_indexes;
	// choose K distinct values for the centers of the clusters
	for(int i = 0; i < K; i++)
	{
		new (&clusters[i]) ClusterStruct(total_values);
		while(true)
		{
			int index_point = rand() % total_points;
			if(find(prohibited_indexes.begin(), prohibited_indexes.end(),
					index_point) == prohibited_indexes.end())
			{
				prohibited_indexes.push_back(index_point);
				points[index_point].id_cluster=i;
				clusters[i].numPoints = 0;
				for(size_t j = 0;j<total_values;j++){
					clusters[i].central_values[j] = points[index_point].values[j];
					clusters[i].central_values_sums[j]=0;
				}
				break;
			}
		}
	}
	for(int i = 0; i < K; i++)
	{
		int total_points_cluster =  clusters[i].numPoints;
		cout << "Cluster " << i + 1 << endl;
		cout << "total_points_cluster " << total_points_cluster << endl;
		cout << "Cluster values: ";
		for(int j = 0; j < total_values; j++)
			cout << clusters[i].central_values[j] << " ";
		cout << endl;
	}
    std::chrono::high_resolution_clock::time_point end_phase1 = chrono::high_resolution_clock::now();
    
	for(int i = 0; i < total_points; i++)
	{
		double sum = 0.0, min_dist;
		int id_cluster_center = 0;
		for(int j = 0; j < total_values; j++)
		{
			sum += pow(clusters[0].central_values[j] -
					   points[i].values[j], 2.0);
		}
		min_dist = sum;
		for(int m = 1; m < K; m++)
		{
			double dist;
			sum = 0.0;
			for(int j = 0; j < total_values; j++)
			{
				dist = clusters[m].central_values[j] -points[i].values[j];
				sum += dist * dist;
			}
    	    //remove the sqrt
			if(sum < min_dist)
			{
				min_dist = sum;
				id_cluster_center = m;
			}
		}
		points[i].id_cluster = id_cluster_center;
	}
	int iter = 2;
	while(true)
	{
		bool done = true;
		
		for(int i = 0; i < K; i++){
			for(size_t j = 0 ; j < total_values;j++){
				clusters[i].central_values_sums[j]=0;
			}
            clusters[i].numPoints = 0;
        }
		//loop_duration = chrono::duration_cast<chrono::microseconds>(chrono::high_resolution_clock::now() - loop1_start);
		//cout << "Loop 1 duration: " << loop_duration.count() << " microseconds" << endl;
		auto loop2_start = chrono::high_resolution_clock::now();
        for(int i = 0; i < total_points; i++){
            for(int j = 0; j < total_values; j++){
                clusters[points[i].id_cluster].central_values_sums[j] += points[i].values[j];
            }
            clusters[points[i].id_cluster].numPoints++;
        }
		
        for(int i = 0; i < K; i++){
            int total_points_cluster = clusters[i].numPoints;
            for(int j = 0; j < total_values; j++){
                double sum = clusters[i].central_values_sums[j];
                clusters[i].central_values[j] = sum / total_points_cluster;
            }
        }
		
		for(int i = 0; i < total_points; i++)
		{
			int id_old_cluster = points[i].id_cluster;
			double sum = 0.0, min_dist;
			int id_cluster_center = 0;
			for(int j = 0; j < total_values; j++)
			{
				sum += pow(clusters[0].central_values[j] -
						   points[i].values[j], 2.0);
			}
			min_dist = sum;
			for(int m = 1; m < K; m++)
			{
				double dist;
				sum = 0.0;
				for(int j = 0; j < total_values; j++)
				{
					dist = clusters[m].central_values[j] -points[i].values[j];
					sum += dist * dist;
				}
    		    //remove the sqrt
				if(sum < min_dist)
				{
					min_dist = sum;
					id_cluster_center = m;
				}
			}
			if(id_old_cluster != id_cluster_center)
			{
				points[i].id_cluster = id_cluster_center;
				done = false;
			}
		}
		
		if(done == true || iter >= max_iterations)
		{
			//cout << "Break in iteration " << iter << "\n\n";
			break;
		}
		iter++;
	}
    std::chrono::high_resolution_clock::time_point end = chrono::high_resolution_clock::now();
	// shows elements of clusters
	for(int i = 0; i < K; i++)
	{
		int total_points_cluster =  clusters[i].numPoints;
		cout << "Cluster " << i + 1 << endl;
		cout << "total_points_cluster " << total_points_cluster << endl;
		cout << "Cluster values: ";
		for(int j = 0; j < total_values; j++)
			cout << clusters[i].central_values[j] << " ";
		cout << endl;
	}
    cout <<std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count() <<endl;
        
	return 0;
}